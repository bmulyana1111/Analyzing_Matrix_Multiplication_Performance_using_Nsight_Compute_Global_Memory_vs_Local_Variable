
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>
#include <math.h>
#include <functional>
#include <stdlib.h> 
#include <time.h>   
#include <iostream>
#include <chrono>
#include <unistd.h>   
#include <fstream>
#include <string>

using namespace std;

#define TILE_SIZE 32

// Matrix multiplication kernel
__global__ void matrixMultiply(float *A, float *B, float *C, int size) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;

    if( row < size && col < size ){
    // do the multiplication for one row and col using local variable 
    for(int k = 0; k < size; k++){
      sum += A[row * size + k] * B[k * size + col];
    }
    // store result
    C[row * size + col] = sum;
  }
  
}


int main() {
    int size = 128;
    int matrixSize = size * size;

    // Allocate memory for matrices on the host
    float *h_A, *h_B, *h_C;
    h_A = (float *)malloc(matrixSize * sizeof(float));
    h_B = (float *)malloc(matrixSize * sizeof(float));
    h_C = (float *)malloc(matrixSize * sizeof(float));

    // Initialize matrices A and B
    for (int i = 0; i < matrixSize; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i);
    }

    // Allocate memory for matrices on the device
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, matrixSize * sizeof(float));
    hipMalloc((void **)&d_B, matrixSize * sizeof(float));
    hipMalloc((void **)&d_C, matrixSize * sizeof(float));

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, h_A, matrixSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize * sizeof(float), hipMemcpyHostToDevice);

    // Set grid and block sizes
    dim3 gridSize(size / TILE_SIZE, size / TILE_SIZE);
    dim3 blockSize(TILE_SIZE, TILE_SIZE);

  auto start = chrono::steady_clock::now();
    // Launch the matrix multiplication kernel
    matrixMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C, size);
  auto end = chrono::steady_clock::now();
  cout << "GPU Elapsed time in nanoseconds: "
        << chrono::duration_cast<chrono::nanoseconds>(end - start).count()
        << " ns" << endl;

    // Copy the result matrix from device to host
    hipMemcpy(h_C, d_C, matrixSize * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

